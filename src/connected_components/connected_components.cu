#include "hip/hip_runtime.h"
#include "pipeline.hh"

#include <map>

namespace gpu
{
  struct Box
  {
    int xmin;
    int ymin;
    int xmax;
    int ymax;

    int size;
  };

  __global__ void gpu_init_label(int* d_in_out, int width, int height)
  {
    int p = blockDim.x * blockIdx.x + threadIdx.x;

    int x = p % width;
    int y = p / width;

    if (x >= width || y >= height)
      return;

    d_in_out[p] *= (p + 2);
  }

  void init_label(int* d_in_out, int width, int height)
  {
    int bsize = 256;
    int g     = std::ceil(((float)(width * height)) / bsize);

    dim3 dimBlock(bsize);
    dim3 dimGrid(g);

    gpu_init_label<<<dimGrid, dimBlock>>>(d_in_out, width, height);
    hipDeviceSynchronize();

    if (hipPeekAtLastError())
      errx(1, "Computation Error");
  }

  namespace one
  {
    __global__ void gpu_propaged_label(int* d_in_out, bool* changed, int width, int height)
    {
      int p = blockDim.x * blockIdx.x + threadIdx.x;

      int x = p % width;
      int y = p / width;

      if (x >= width || y >= height || d_in_out[p] == 0)
        return;

      int cmin = d_in_out[p];

      int min = d_in_out[p];
      for (int j = -1; j < 2; j++)
      {
        int cy = y + j;
        if (!(0 <= cy && cy < height))
          continue;

        for (int i = -1; i < 2; i++)
        {
          int cx = x + i;
          if (!(0 <= cx && cx < width))
            continue;

          int pos  = cx + cy * width;
          int cpos = d_in_out[pos];
          if (cpos && cpos < min)
            min = cpos;
        }
      }

      if (min < cmin)
      {
        *changed    = true;
        d_in_out[p] = min;
      }
    }

    void propaged_label(int* d_in_out, bool* d_changed, int width, int height)
    {
      int bsize = 256;
      int g     = std::ceil(((float)(width * height)) / bsize);

      dim3 dimBlock(bsize);
      dim3 dimGrid(g);

      gpu_propaged_label<<<dimGrid, dimBlock>>>(d_in_out, d_changed, width, height);
      hipDeviceSynchronize();

      if (hipPeekAtLastError())
        errx(1, "Computation Error");
    }

    std::set<std::vector<int>> get_connected_components(int* d_in_out, int width, int height, int minimum_pixel)
    {
      init_label(d_in_out, width, height);

      bool  changed   = true;
      bool* h_changed = &changed;

      bool* d_changed;
      int   rc = hipMalloc(&d_changed, sizeof(bool));
      if (rc)
        errx(1, "Fail buffer allocation for d_changed");

      while (changed)
      {
        changed = false;
        rc      = hipMemcpy(d_changed, h_changed, sizeof(bool), hipMemcpyHostToDevice);
        if (rc)
          errx(1, "Fail buffer copy to device");

        propaged_label(d_in_out, d_changed, width, height);

        rc = hipMemcpy(h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);
        if (rc)
          errx(1, "Fail buffer copy to host");
      }

      int* h = static_cast<int*>(std::malloc(sizeof(int) * width * height));
      rc     = hipMemcpy(h, d_in_out, sizeof(int) * width * height, hipMemcpyDeviceToHost);
      if (rc)
        errx(1, "Fail buffer copy to host");

      auto ret = cpu::compute_find(h, width, height, minimum_pixel, false);

      std::free(h);

      return ret;
    }
  } // namespace one

  namespace two
  {
    __global__ void gpu_propaged_label(int* d_in, int* d_out, bool* changed, int width, int height)
    {
      int p = blockDim.x * blockIdx.x + threadIdx.x;

      int x = p % width;
      int y = p / width;

      if (x >= width || y >= height || d_in[p] == 0)
        return;

      int cmin = d_in[p];

      int min = d_in[p];
      for (int j = -1; j < 2; j++)
      {
        int cy = y + j;
        if (!(0 <= cy && cy < height))
          continue;

        for (int i = -1; i < 2; i++)
        {
          int cx = x + i;
          if (!(0 <= cx && cx < width))
            continue;

          int pos  = cx + cy * width;
          int cpos = d_in[pos];
          if (cpos && cpos < min)
            min = cpos;
        }
      }

      if (min < cmin)
      {
        *changed = true;
        cmin     = min;
      }

      d_out[p] = cmin;
    }

    void propaged_label(int* d_in, int* d_out, bool* d_changed, int width, int height)
    {
      int bsize = 256;
      int g     = std::ceil(((float)(width * height)) / bsize);

      dim3 dimBlock(bsize);
      dim3 dimGrid(g);

      gpu_propaged_label<<<dimGrid, dimBlock>>>(d_in, d_out, d_changed, width, height);
      hipDeviceSynchronize();

      if (hipPeekAtLastError())
        errx(1, "Computation Error");
    }

    void swap(int** a, int** b)
    {
      auto tmp = *a;
      *b       = *a;
      *a       = tmp;
    }

    std::set<std::vector<int>> get_connected_components(int* d_A, int* d_B, int* h, int width, int height,
                                                        int minimum_pixel)
    {
      init_label(d_A, width, height);
      hipMemset((void*)d_B, 0, sizeof(int) * width * height);

      bool  changed   = true;
      bool* h_changed = &changed;

      bool* d_changed;
      int   rc = hipMalloc(&d_changed, sizeof(bool));
      if (rc)
        errx(1, "Fail changed allocation");

      while (changed)
      {
        changed = false;
        rc      = hipMemcpy(d_changed, h_changed, sizeof(bool), hipMemcpyHostToDevice);
        if (rc)
          errx(1, "Fail buffer copy to device");

        propaged_label(d_A, d_B, d_changed, width, height);
        swap(&d_A, &d_B);

        rc = hipMemcpy(h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);
        if (rc)
          errx(1, "Fail buffer copy to host");
      }

      rc = hipMemcpy(h, d_A, sizeof(int) * width * height, hipMemcpyDeviceToHost);
      if (rc)
        errx(1, "Fail buffer copy to host");

      auto ret = cpu::compute_find(h, width, height, minimum_pixel, false);

      return ret;
    }
  } // namespace two
} // namespace gpu