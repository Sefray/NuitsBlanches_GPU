#include "hip/hip_runtime.h"
#include "pipeline.cuh"

namespace gpu
{
  __global__ void gpu_difference(int* d_ref_in, int* d_in, int* d_out, int width, int height)
  {
    int p = blockDim.x * blockIdx.x + threadIdx.x;

    int x = p % width;
    int y = p / width;

    if (x >= width || y >= height)
      return;

    int c    = x + y * width;
    d_out[c] = std::abs(d_in[c] - d_ref_in[c]);
  }

  namespace one
  {
    int* compute_difference(int* d_ref_in, int* d_in, int width, int height)
    {
      int* d_out = my_cuda_malloc(sizeof(int) * width * height);

      int bsize = 256;
      int g     = std::ceil(((float)(width * height)) / bsize);

      dim3 dimBlock(bsize);
      dim3 dimGrid(g);

      gpu_difference<<<dimGrid, dimBlock>>>(d_ref_in, d_in, d_out, width, height);

      if (hipPeekAtLastError())
        errx(1, "Computation Error");

      hipFree(d_in);

      return d_out;
    }
  } // namespace one

  namespace one::two
  {
    void compute_difference(int* d_ref_in, int* d_in, int* d_out, int width, int height)
    {
      int bsize = 256;
      int g     = std::ceil(((float)(width * height)) / bsize);

      dim3 dimBlock(bsize);
      dim3 dimGrid(g);

      gpu_difference<<<dimGrid, dimBlock>>>(d_ref_in, d_in, d_out, width, height);

      if (hipPeekAtLastError())
        errx(1, "Computation Error");
    }
  } // namespace one::two
} // namespace gpu