#include "pipeline.hh"
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <numbers>
#include <set>

#include "debug/debug.hh"
#include <err.h>

namespace gpu
{
  void my_cuda_mem_copy(const int* h, int* d, size_t n)
  {
    int rc = hipMemcpy(d, h, n, hipMemcpyHostToDevice);
    if (rc)
      errx(1, "Fail buffer copy to device");
  }

  void my_cuda_mem_set(int* d, int v, size_t n)
  {
    int rc = hipMemset((void*)d, 0, n);
    if (rc)
      errx(1, "Fail buffer copy to device");
  }

  int* malloc_and_copy(const int* h, int width, int height)
  {
    int* d = my_cuda_malloc(sizeof(int) * width * height);
    my_cuda_mem_copy(h, d, sizeof(int) * width * height);
    return d;
  }

  void my_cuda_free(int* d)
  {
    hipError_t rc = hipSuccess;
    rc             = hipFree(d);
    if (rc)
      errx(1, "Fail to free memory");
  }

  int* my_cuda_malloc(size_t n)
  {
    int* d_out;
    int  rc = hipMalloc(&d_out, n);
    if (rc)
      errx(1, "Fail buffer allocation for d_out");
    return d_out;
  }

  int* my_cuda_calloc(size_t n)
  {
    int* d_out = my_cuda_malloc(n);
    my_cuda_mem_set(d_out, 0, n);
    return d_out;
  }

  namespace one
  {
    std::set<std::vector<int>> pipeline(int* d_ref_in, unsigned char* h_input, int width, int height, int kernel_size,
                                        int kernel_size_opening, int kernel_size_closing, int binary_threshold,
                                        enum mode_cc mode_cc, int minimum_pixel)
    {
      unsigned char* d_input;

      int rc = hipMalloc(&d_input, sizeof(unsigned char) * width * height * 3);
      if (rc)
        errx(1, "Error in buffer_uc allocation");
      rc = hipMemcpy(d_input, h_input, sizeof(unsigned char) * width * height * 3, hipMemcpyHostToDevice);
      if (rc)
        errx(1, "Error in buffer_uc copy");

      // 1.Greyscale
      auto d_greyscale = greyscale(d_input, width, height);

      // 2.Smooth (gaussian filter)
      auto d_smoothed = smoothing(d_greyscale, width, height, kernel_size);

      // 3.Difference
      auto d_diff = compute_difference(d_ref_in, d_smoothed, width, height);

      // 4.Closing/opening with disk or rectangle
      auto d_closed_opened = closing_opening(d_diff, width, height, kernel_size_opening, kernel_size_closing);

      // 5.1.Thresh image
      binary_image(d_closed_opened, width, height, binary_threshold);

      // 5.2.Lakes
      auto components = get_connected_components(d_closed_opened, width, height, minimum_pixel);

      hipFree(d_closed_opened);

      return components;
    }
  } // namespace one

  namespace one::two
  {
    std::set<std::vector<int>> pipeline(int* d_ref_in, unsigned char* h_input, int width, int height, int kernel_size,
                                        int kernel_size_opening, int kernel_size_closing, int binary_threshold,
                                        enum mode_cc mode_cc, int minimum_pixel, unsigned char* d_buffer_uc,
                                        int* d_buffer_A, int* d_buffer_B)
    {
      hipMemcpy(d_buffer_uc, h_input, sizeof(unsigned char) * width * height * 3, hipMemcpyHostToDevice);

      // 1.Greyscale
      greyscale(d_buffer_uc, d_buffer_A, width, height);

      // Buffer Allocation
      my_cuda_mem_set(d_buffer_B, 0, sizeof(int) * width * height);

      // 2.Smooth (gaussian filter)
      smoothing(d_buffer_A, d_buffer_B, width, height, kernel_size);

      // 3.Difference
      compute_difference(d_ref_in, d_buffer_B, d_buffer_A, width, height);

      // 4.Closing/opening with disk or rectangle
      closing_opening(d_buffer_A, d_buffer_B, width, height, kernel_size_opening, kernel_size_closing);

      // 5.1.Thresh image
      binary_image(d_buffer_A, width, height, binary_threshold);

      // 5.2.Lakes
      auto components = get_connected_components(d_buffer_A, d_buffer_B, width, height, minimum_pixel);

      return components;
    }
  } // namespace one::two

  namespace one::two::three
  {
    std::set<std::vector<int>> pipeline(int* d_ref_in, unsigned char* h_input, int width, int height, int kernel_size,
                                        int kernel_size_opening, int kernel_size_closing, int binary_threshold,
                                        enum mode_cc mode_cc, int minimum_pixel, unsigned char* d_buffer_uc,
                                        int* d_buffer_A, int* d_buffer_B)
    {
      hipMemcpy(d_buffer_uc, h_input, sizeof(unsigned char) * width * height * 3, hipMemcpyHostToDevice);

      // 1.Greyscale
      greyscale(d_buffer_uc, d_buffer_A, width, height);

      // Buffer Allocation
      my_cuda_mem_set(d_buffer_B, 0, sizeof(int) * width * height);

      // 2.Smooth (gaussian filter)
      smoothing(d_buffer_A, d_buffer_B, width, height, kernel_size);

      // 3.Difference
      compute_difference(d_ref_in, d_buffer_B, d_buffer_A, width, height);

      // 4.Closing/opening with disk or rectangle
      three::closing_opening(d_buffer_A, d_buffer_B, width, height, kernel_size_opening, kernel_size_closing);

      // 5.1.Thresh image
      binary_image(d_buffer_A, width, height, binary_threshold);

      // 5.2.Lakes
      auto components = get_connected_components(d_buffer_A, d_buffer_B, width, height, minimum_pixel);

      return components;
    }
  } // namespace one::two::three

  namespace one::two::three::four
  {
    std::set<std::vector<int>> pipeline(int* d_ref_in, unsigned char* h_input, int width, int height, int kernel_size,
                                        int kernel_size_opening, int kernel_size_closing, int binary_threshold,
                                        enum mode_cc mode_cc, int minimum_pixel, unsigned char* d_buffer_uc,
                                        int* d_buffer_A, int* d_buffer_B)
    {
      hipMemcpy(d_buffer_uc, h_input, sizeof(unsigned char) * width * height * 3, hipMemcpyHostToDevice);

      // 1.Greyscale
      greyscale(d_buffer_uc, d_buffer_A, width, height);

      // Buffer Allocation
      my_cuda_mem_set(d_buffer_B, 0, sizeof(int) * width * height);

      // 2.Smooth (gaussian filter)
      smoothing(d_buffer_A, d_buffer_B, width, height, kernel_size);

      // 3.Difference
      compute_difference(d_ref_in, d_buffer_B, d_buffer_A, width, height);

      // 4.Closing/opening with disk or rectangle
      closing_opening(d_buffer_A, d_buffer_B, width, height, kernel_size_opening, kernel_size_closing);

      // 5.1.Thresh image
      binary_image(d_buffer_A, width, height, binary_threshold);

      // 5.2.Lakes
      auto components = get_connected_components(d_buffer_A, d_buffer_B, width, height, minimum_pixel);

      return components;
    }
  } // namespace one::two::three::four
} // namespace gpu