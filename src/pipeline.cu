#include "pipeline.hh"
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <numbers>
#include <set>

#include "debug/debug.hh"
#include <err.h>

namespace gpu
{
        int *malloc_and_copy(const int *h, int width, int height)
        {
                hipError_t rc = hipSuccess;

                int *d;

                rc = hipMalloc((void **)&d, sizeof(int) * width * height);
                if (rc)
                        errx(1, "Fail buffer allocation");

                hipMemcpy(d, h, sizeof(int) * width * height, hipMemcpyHostToDevice);
                if (rc)
                        errx(1, "Fail buffer copy to device");

                return d;
        }

        void my_cuda_free(int *d)
        {
                hipError_t rc = hipSuccess;
                rc = hipFree(d);
                if (rc)
                        errx(1, "Fail to free memory");
        }

        std::set<std::vector<int>> pipeline(int *d_ref_in, png::pixel_buffer<png::rgb_pixel> h_input, int width, int height, int kernel_size, int kernel_size_opening, int kernel_size_closing, int binary_threshold, enum mode_cc mode_cc, int minimum_pixel)
        {
                hipError_t rc = hipSuccess;

                // 1.Greyscale
                auto h_greyscale = cpu::greyscale(h_input, width, height);

                // Buffer Allocation
                int *d_buffer_A;
                rc = hipMalloc(&d_buffer_A, sizeof(int) * width * height);
                if (rc)
                        errx(1, "Fail buffer allocation for A");

                int *d_buffer_B;
                rc = hipMalloc(&d_buffer_B, sizeof(int) * width * height);
                if (rc)
                        errx(1, "Fail buffer allocation for B");
                hipMemset(d_buffer_B, 0, sizeof(int) * width * height);

                rc = hipMemcpy(d_buffer_A, h_greyscale, sizeof(int) * width * height, hipMemcpyHostToDevice);
                if (rc)
                        errx(1, "Fail buffer copy to device");

                // 2.Smooth (gaussian filter)
                smoothing(d_buffer_A, d_buffer_B, width, height, kernel_size);

#ifndef NDEBUG
                rc = hipMemcpy(h_greyscale, d_buffer_B, sizeof(int) * width * height, hipMemcpyDeviceToHost);
                if (rc)
                        errx(1, "Fail buffer copy to host");
                save_img(h_greyscale, width, height, "gpu_smoothed.png");
#endif

                // 3.Difference
                compute_difference(d_ref_in, d_buffer_B, d_buffer_A, width, height);

#ifndef NDEBUG
                rc = hipMemcpy(h_greyscale, d_buffer_A, sizeof(int) * width * height, hipMemcpyDeviceToHost);
                if (rc)
                        errx(1, "Fail buffer copy to host");
                save_img(h_greyscale, width, height, "gpu_diff.png");
#endif

                // 4.Closing/opening with disk or rectangle
                closing_opening(d_buffer_A, d_buffer_B, width, height, kernel_size_opening, kernel_size_closing);
#ifndef NDEBUG
                rc = hipMemcpy(h_greyscale, d_buffer_A, sizeof(int) * width * height, hipMemcpyDeviceToHost);
                if (rc)
                        errx(1, "Fail buffer copy to host");
                save_img(h_greyscale, width, height, "gpu_closing_opening.png");
#endif

                // 5.1.Thresh image
                binary_image(d_buffer_A, width, height, binary_threshold);
#ifndef NDEBUG
                rc = hipMemcpy(h_greyscale, d_buffer_A, sizeof(int) * width * height, hipMemcpyDeviceToHost);
                if (rc)
                        errx(1, "Fail buffer copy to host");
                save_img(h_greyscale, width, height, "gpu_binary.png", 255);
#endif
                rc = hipMemcpy(h_greyscale, d_buffer_A, sizeof(int) * width * height, hipMemcpyDeviceToHost);
                if (rc)
                        errx(1, "Fail buffer copy to host");
                rc = hipFree(d_buffer_A);
                if (rc)
                        errx(1, "Fail to free memory");
                rc = hipFree(d_buffer_B);
                if (rc)
                        errx(1, "Fail to free memory");

                // 5.2.Lakes
                auto components = cpu::get_connected_components(h_greyscale, width, height, mode_cc, minimum_pixel);

                // TMP
                std::free(h_greyscale);

                return components;
        }
}